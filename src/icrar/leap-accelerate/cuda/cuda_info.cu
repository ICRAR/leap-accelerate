/**
 * ICRAR - International Centre for Radio Astronomy Research
 * (c) UWA - The University of Western Australia
 * Copyright by UWA(in the framework of the ICRAR)
 * All rights reserved
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License along
 * with this program; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include <hip/hip_runtime.h>

#include "cuda_info.h"
#include "helper_cuda.cuh"


#include <iostream>

int GetCudaDeviceCount()
{
    int deviceCount = 0;
    checkCudaErrors(hipGetDeviceCount(&deviceCount));
    return deviceCount;
}

void printCudaVersion()
{
#ifdef __NVCC__
    std::cout << "CUDA NVCC Compiler version: " << __CUDACC_VER_MAJOR__ << __CUDACC_VER_MINOR__ << __CUDACC_VER_BUILD__ << std::endl;
#endif

    int runtime_ver = 0;
    hipRuntimeGetVersion(&runtime_ver);
    std::cout << "CUDA Runtime version: " << runtime_ver << std::endl;

    int driver_ver = 0;
    hipDriverGetVersion(&driver_ver);
    std::cout << "CUDA Driver version: " << driver_ver << std::endl;
}