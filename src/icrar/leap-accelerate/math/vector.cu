#include "hip/hip_runtime.h"
/**
*    ICRAR - International Centre for Radio Astronomy Research
*    (c) UWA - The University of Western Australia
*    Copyright by UWA (in the framework of the ICRAR)
*    All rights reserved
*
*    This library is free software; you can redistribute it and/or
*    modify it under the terms of the GNU Lesser General Public
*    License as published by the Free Software Foundation; either
*    version 2.1 of the License, or (at your option) any later version.
*
*    This library is distributed in the hope that it will be useful,
*    but WITHOUT ANY WARRANTY; without even the implied warranty of
*    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
*    Lesser General Public License for more details.
*
*    You should have received a copy of the GNU Lesser General Public
*    License along with this library; if not, write to the Free Software
*    Foundation, Inc., 59 Temple Place, Suite 330, Boston,
*    MA 02111-1307  USA
*/

#include "vector.h"
#include <icrar/leap-accelerate/math/vector.cuh>

void printCudaVersion()
{
    std::cout << "CUDA Compiled version: " << __CUDACC_VER__ << std::endl;

    int runtime_ver;
    hipRuntimeGetVersion(&runtime_ver);
    std::cout << "CUDA Runtime version: " << runtime_ver << std::endl;

    int driver_ver;
    hipDriverGetVersion(&driver_ver);
    std::cout << "CUDA Driver version: " << driver_ver << std::endl;
}

void h_add(const casacore::Array<double>& a, const casacore::Array<double>& b, casacore::Array<double>& c)
{
   h_add(a, b, c);
}

// __global__ void h_add(const int* x1, const int* x2, int* y)
// {
//     d_add(x1, x2, y);
// }

// __global__ void h_add(const float* x1, const float* x2, float* y)
// {
//    d_add(x1, x2, y);
// }

// __global__ void h_add(const double* x1, const double* x2, double* y)
// {
//    d_add(x1, x2, y);
// }

// extern "C"
// {
//    __global__ void addi(const int* x1, const int* x2, int* y);

//    __global__ void addf(const float* x1, const float* x2, float* y);

//    __global__ void addd(const double* x1, const double* x2, double* y);
// }