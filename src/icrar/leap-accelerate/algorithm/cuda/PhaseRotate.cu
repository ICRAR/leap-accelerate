#include "hip/hip_runtime.h"

/**
 * ICRAR - International Centre for Radio Astronomy Research
 * (c) UWA - The University of Western Australia
 * Copyright by UWA(in the framework of the ICRAR)
 * All rights reserved
 *
 * This library is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation; either
 * version 2.1 of the License, or (at your option) any later version.
 *
 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with this library; if not, write to the Free Software
 * Foundation, Inc., 59 Temple Place, Suite 330, Boston,
 * MA 02111 - 1307  USA
 */

#include "PhaseRotate.h"

#include <icrar/leap-accelerate/common/Tensor3X.h>
#include <icrar/leap-accelerate/math/casacore_helper.h>
#include <icrar/leap-accelerate/math/vector_extensions.h>

#include <icrar/leap-accelerate/model/cpu/Integration.h>
#include <icrar/leap-accelerate/model/cuda/DeviceMetaData.h>
#include <icrar/leap-accelerate/model/cuda/DeviceIntegration.h>

#include <icrar/leap-accelerate/math/cuda/math.cuh>
#include <icrar/leap-accelerate/math/cuda/matrix.h>
#include <icrar/leap-accelerate/math/cuda/vector.h>
#include <icrar/leap-accelerate/math/cpu/vector.h>
#include <icrar/leap-accelerate/cuda/cuda_info.h>
#include <icrar/leap-accelerate/core/log/logging.h>
#include <icrar/leap-accelerate/core/profiling/timer.h>

#include <icrar/leap-accelerate/common/eigen_extensions.h>

#include <casacore/measures/Measures/MDirection.h>
#include <casacore/casa/Quanta/MVDirection.h>
#include <casacore/casa/Quanta/MVuvw.h>
#include <casacore/casa/Arrays/Matrix.h>
#include <casacore/casa/Arrays/Vector.h>

#include <boost/math/constants/constants.hpp>

#include <hip/hip_complex.h>
#include <hip/hip_math_constants.h>

#include <complex>
#include <istream>
#include <iostream>
#include <iterator>
#include <string>
#include <queue>
#include <exception>
#include <memory>
#include <set>

using Radians = double;
using namespace boost::math::constants;

namespace icrar
{
namespace cuda
{
    cpu::CalibrateResult Calibrate(
        const icrar::MeasurementSet& ms,
        const std::vector<icrar::MVDirection>& directions,
        double minimumBaselineThreshold,
        bool isFileSystemCacheEnabled)
    {
        LOG(info) << "Starting Calibration using cuda";
        LOG(info)
        << "stations: " << ms.GetNumStations() << ", "
        << "rows: " << ms.GetNumRows() << ", "
        << "baselines: " << ms.GetNumBaselines() << ", "
        << "flagged baselines: " << ms.GetNumFlaggedBaselines() << ", "
        << "baseline threshold: " << minimumBaselineThreshold << ", "
        << "short baselines: " << ms.GetNumShortBaselines(minimumBaselineThreshold) << ", "
        << "filtered baselines: " << ms.GetNumFilteredBaselines(minimumBaselineThreshold) << ", "
        << "channels: " << ms.GetNumChannels() << ", "
        << "polarizations: " << ms.GetNumPols() << ", "
        << "directions: " << directions.size() << ", "
        << "timesteps: " << ms.GetNumRows() / ms.GetNumBaselines();

        profiling::timer calibration_timer;

        if(GetCudaDeviceCount() == 0)
        {
            throw std::runtime_error("Could not find CUDA device");
        }

        profiling::timer integration_read_timer;
        auto output_integrations = std::vector<std::vector<cpu::IntegrationResult>>();
        auto output_calibrations = std::vector<std::vector<cpu::CalibrationResult>>();
        auto input_queue = std::vector<cuda::DeviceIntegration>();

        // Flooring to remove incomplete measurements
        int integrations = ms.GetNumRows() / ms.GetNumBaselines();
        if(integrations == 0)
        {
            std::stringstream ss;
            ss << "invalid number of rows, expected >" << ms.GetNumBaselines() << ", got " << ms.GetNumRows();
            throw icrar::file_exception(ms.GetFilepath().get_value_or("unknown"), ss.str(), __FILE__, __LINE__);
        }

        auto integration = cpu::Integration(
            0,
            ms,
            0,
            ms.GetNumChannels(),
            ms.GetNumRows(),
            ms.GetNumPols());

        for(int i = 0; i < directions.size(); ++i)
        {                
            output_integrations.emplace_back();
            output_calibrations.emplace_back();
        }
        LOG(info) << "Read integration data in " << integration_read_timer;

        profiling::timer metadata_read_timer;
        LOG(info) << "Loading MetaData";
        
        auto metadata = icrar::cpu::MetaData(ms, integration.GetUVW(), minimumBaselineThreshold, isFileSystemCacheEnabled);
        
        auto constantBuffer = std::make_shared<ConstantBuffer>(
            metadata.GetConstants(),
            metadata.GetA(),
            metadata.GetI(),
            metadata.GetAd(),
            metadata.GetA1(),
            metadata.GetI1(),
            metadata.GetAd1()
        );

        auto solutionIntervalBuffer = std::make_shared<SolutionIntervalBuffer>(metadata.GetOldUVW());
        
        auto directionBuffer = std::make_shared<DirectionBuffer>(
            metadata.GetDirection(),
            metadata.GetDD(),
            metadata.GetOldUVW().size(),
            metadata.GetAvgData().rows(),
            metadata.GetAvgData().cols());

        auto deviceMetadata = icrar::cuda::DeviceMetaData(constantBuffer, solutionIntervalBuffer, directionBuffer);

        // Emplace a single empty tensor
        input_queue.emplace_back(0, integration.GetVis().dimensions());
        
        LOG(info) << "Metadata loaded in " << metadata_read_timer;

        profiling::timer phase_rotate_timer;
        for(int i = 0; i < directions.size(); ++i)
        {
            LOG(info) << "Processing direction " << i;
            LOG(info) << "Setting Metadata";
            metadata.SetDirection(directions[i]);

            directionBuffer->SetDirection(metadata.GetDirection());
            directionBuffer->SetDD(metadata.GetDD());
            directionBuffer->GetAvgData().SetZeroSync();

            input_queue[0].SetData(integration);

            LOG(info) << "Copying Metadata to Device";
            LOG(info) << "PhaseRotate";

            icrar::cuda::RotateUVW(
                deviceMetadata.GetDD(),
                solutionIntervalBuffer->GetOldUVW(),
                directionBuffer->GetUVW());

            icrar::cuda::PhaseRotate(
                metadata,
                deviceMetadata,
                directions[i],
                input_queue,
                output_integrations[i],
                output_calibrations[i]);
        }
        LOG(info) << "Performed PhaseRotate in " << phase_rotate_timer;

        LOG(info) << "Finished calibration in " << calibration_timer;
        return std::make_pair(std::move(output_integrations), std::move(output_calibrations));
    }

    __global__ void g_RotateUVW(
        Eigen::Matrix3d dd,
        const double* pOldUVW,
        double* pUVW,
        int uvwLength)
    {
        auto oldUVWs = Eigen::Map<const Eigen::Matrix<double, Eigen::Dynamic, 3, Eigen::RowMajor>>(pOldUVW, uvwLength, 3);
        auto UVWs = Eigen::Map<Eigen::Matrix<double, Eigen::Dynamic, 3, Eigen::RowMajor>>(pUVW, uvwLength, 3);
        int row = blockDim.x * blockIdx.x + threadIdx.x;
        auto oldUvw = Eigen::RowVector3d(oldUVWs(row, 0), oldUVWs(row, 1), oldUVWs(row, 2));
        Eigen::RowVector3d uvw = oldUvw * dd;
        UVWs(row, 0) = uvw(0);
        UVWs(row, 1) = uvw(1);
        UVWs(row, 2) = uvw(2);
    }

    __host__ void RotateUVW(Eigen::Matrix3d dd, const device_vector<icrar::MVuvw>& oldUVW, device_vector<icrar::MVuvw>& UVW)
    {
        assert(oldUVW.GetCount() != UVW.GetCount());
        dim3 blockSize = dim3(1024, 1, 1);
        dim3 gridSize = dim3((int)ceil((float)oldUVW.GetCount() / blockSize.x), 1, 1);
        g_RotateUVW<<<blockSize, gridSize>>>(dd, oldUVW.Get()->data(), UVW.Get()->data(), oldUVW.GetCount());
    }

    void PhaseRotate(
        cpu::MetaData& metadata,
        DeviceMetaData& deviceMetadata,
        const icrar::MVDirection& direction,
        std::vector<cuda::DeviceIntegration>& input,
        std::vector<cpu::IntegrationResult>& output_integrations,
        std::vector<cpu::CalibrationResult>& output_calibrations)
    {
        for(auto& integration : input)
        {
            LOG(info) << "Rotating integration " << integration.GetIntegrationNumber();
            icrar::cuda::RotateVisibilities(integration, deviceMetadata);

            //TODO: currently unused
            output_integrations.emplace_back(
                integration.GetIntegrationNumber(),
                direction,
                boost::optional<std::vector<Eigen::VectorXd>>());
        }

        LOG(info) << "Copying Metadata from Device";
        deviceMetadata.AvgDataToHost(metadata.GetAvgData());

        LOG(info) << "Calibrating on cpu";
        trace_matrix(metadata.GetAvgData(), "avg_data");

        auto phaseAngles = icrar::arg(metadata.GetAvgData());
        
        // PhaseAngles I1
        // Value at last index of phaseAnglesI1 must be 0 (which is the reference antenna phase value)
        Eigen::VectorXd phaseAnglesI1 = icrar::cpu::VectorRangeSelect(phaseAngles, metadata.GetI1(), 0); // 1st pol only
        phaseAnglesI1.conservativeResize(phaseAnglesI1.rows() + 1);
        phaseAnglesI1(phaseAnglesI1.rows() - 1) = 0;

        Eigen::VectorXd cal1 = metadata.GetAd1() * phaseAnglesI1;
        
        Eigen::MatrixXd dInt = Eigen::MatrixXd::Zero(metadata.GetI().size(), metadata.GetAvgData().cols());
        
        for(int n = 0; n < metadata.GetI().size(); ++n)
        {
            double sum = metadata.GetA()(n, Eigen::all) * cal1;
            dInt(n, Eigen::all) = icrar::arg(std::exp(std::complex<double>(0, -sum * two_pi<double>())) * metadata.GetAvgData()(n, Eigen::all));
        }

        Eigen::VectorXd deltaPhaseColumn = dInt(Eigen::all, 0); // 1st pol only
        deltaPhaseColumn.conservativeResize(deltaPhaseColumn.size() + 1);
        deltaPhaseColumn(deltaPhaseColumn.size() - 1) = 0;
        output_calibrations.emplace_back(direction, (metadata.GetAd() * deltaPhaseColumn) + cal1);
    }

    /**
     * @brief Rotates visibilities in parallel for baselines and channels
     * @note Atomic operator required for writing to @p pAvgData
     */
    __global__ void g_RotateVisibilities(
        hipDoubleComplex* pIntegrationData, int integration_data_dim0, int integration_data_dim1, int integration_data_dim2,
        icrar::cpu::Constants constants,
        Eigen::Matrix3d dd, //TODO(cgray) remove
        double2 direction, //TODO(cgray) remove
        double3* uvw, int uvwLength,
        double3* oldUVW, int oldUVWLegth,
        hipDoubleComplex* pAvgData, int avgDataRows, int avgDataCols)
    {
        using Tensor2Xcucd = Eigen::Tensor<hipDoubleComplex, 2>;
        using Tensor3Xcucd = Eigen::Tensor<hipDoubleComplex, 3>;
        
        const int integration_baselines = integration_data_dim1;
        const int integration_channels = integration_data_dim2;
        const int md_baselines = constants.nbaselines; //metadata baselines
        const int polarizations = constants.num_pols;

        //parallel execution per channel
        int baseline = blockDim.x * blockIdx.x + threadIdx.x;
        int channel = blockDim.y * blockIdx.y + threadIdx.y;

        if(baseline < integration_baselines && channel < integration_channels)
        {
            auto integration_data = Eigen::TensorMap<Tensor3Xcucd>(pIntegrationData, integration_data_dim0, integration_data_dim1, integration_data_dim2);
            auto avg_data = Eigen::TensorMap<Tensor2Xcucd>(pAvgData, avgDataRows, avgDataCols);
    
            int md_baseline = baseline % md_baselines;

            // loop over baselines
            constexpr double two_pi = 2 * HIP_PI;
            double shiftFactor = two_pi * (uvw[baseline].z - oldUVW[baseline].z);

            // loop over channels
            double shiftRad = shiftFactor / constants.GetChannelWavelength(channel);

            hipDoubleComplex exp = cuCexp(make_hipDoubleComplex(0.0, shiftRad));

            for(int polarization = 0; polarization < polarizations; polarization++)
            {
                 integration_data(polarization, baseline, channel) = hipCmul(integration_data(polarization, baseline, channel), exp);
            }

            bool hasNaN = false;
            for(int polarization = 0; polarization < polarizations; polarization++)
            {
                auto n = integration_data(polarization, baseline, channel);
                hasNaN |= isnan(n.x) || isnan(n.y);
            }

            if(!hasNaN)
            {
                for(int polarization = 0; polarization < polarizations; ++polarization)
                {
                    atomicAdd(&avg_data(md_baseline, polarization).x, integration_data(polarization, baseline, channel).x);
                    atomicAdd(&avg_data(md_baseline, polarization).y, integration_data(polarization, baseline, channel).y);
                }
            }
        }
    }

    __host__ void RotateVisibilities(
        DeviceIntegration& integration,
        DeviceMetaData& metadata)
    {
        const auto& constants = metadata.GetConstants(); 
        assert(constants.channels == integration.GetChannels() && integration.GetChannels() == integration.GetVis().GetDimensionSize(2));
        assert(constants.nbaselines == metadata.GetAvgData().GetRows() && integration.GetBaselines() == integration.GetVis().GetDimensionSize(1));
        assert(constants.num_pols == integration.GetVis().GetDimensionSize(0));

        // block size can any value where the product is 1024
        dim3 blockSize = dim3(128, 8, 1);
        dim3 gridSize = dim3(
            (int)ceil((float)integration.GetBaselines() / blockSize.x),
            (int)ceil((float)integration.GetChannels() / blockSize.y),
            1
        );

        //TODO: store polar form in advance
        const auto polar_direction = icrar::ToPolar(metadata.GetDirection());
        g_RotateVisibilities<<<gridSize, blockSize>>>(
            (hipDoubleComplex*)integration.GetVis().Get(), integration.GetVis().GetDimensionSize(0), integration.GetVis().GetDimensionSize(1), integration.GetVis().GetDimensionSize(2),
            constants,
            metadata.GetDD(),
            make_double2(polar_direction(0), polar_direction(1)),
            (double3*)metadata.GetUVW().Get(), metadata.GetUVW().GetCount(),
            (double3*)metadata.GetOldUVW().Get(), metadata.GetOldUVW().GetCount(),
            (hipDoubleComplex*)metadata.GetAvgData().Get(), metadata.GetAvgData().GetRows(), metadata.GetAvgData().GetCols());
    }
}
}
