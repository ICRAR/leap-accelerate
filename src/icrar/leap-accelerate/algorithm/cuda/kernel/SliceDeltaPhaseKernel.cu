#include "hip/hip_runtime.h"
/**
 * ICRAR - International Centre for Radio Astronomy Research
 * (c) UWA - The University of Western Australia
 * Copyright by UWA(in the framework of the ICRAR)
 * All rights reserved
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License along
 * with this program; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include "SliceDeltaPhaseKernel.h"
#include <icrar/leap-accelerate/math/cpu/math.h>
#include <icrar/leap-accelerate/exception/exception.h>

namespace icrar
{
namespace cuda
{
    __global__ void g_SliceDeltaPhase(
        const Eigen::Map<const Eigen::MatrixXd> deltaPhase,
        Eigen::Map<Eigen::VectorXd> deltaPhaseColumn);

    __host__ void SliceDeltaPhase(
        const device_matrix<double>& deltaPhase,
        device_vector<double>& deltaPhaseColumn)
    {
        if(deltaPhase.GetRows()+1 != deltaPhaseColumn.GetRows())
        {
            throw invalid_argument_exception("incorrect number of columns", "deltaPhaseColumn", __FILE__, __LINE__);
        }
        auto deltaPhaseMap = Eigen::Map<const Eigen::MatrixXd>(deltaPhase.Get(), deltaPhase.GetRows(), deltaPhase.GetCols());
        auto deltaPhaseColumnMap = Eigen::Map<Eigen::VectorXd>(deltaPhaseColumn.Get(), deltaPhaseColumn.GetRows());
        
        dim3 blockSize = dim3(1024, 1, 1);
        dim3 gridSize = dim3(cpu::ceil_div<int64_t>(deltaPhaseColumn.GetRows(),  blockSize.x), 1, 1);
        g_SliceDeltaPhase<<<blockSize,gridSize>>>(deltaPhaseMap, deltaPhaseColumnMap);
    }

    __global__ void g_SliceDeltaPhase(
        const Eigen::Map<const Eigen::MatrixXd> deltaPhase,
        Eigen::Map<Eigen::VectorXd> deltaPhaseColumn)
    {
        int row = blockDim.x * blockIdx.x + threadIdx.x;
        if(row < deltaPhase.rows())
        {
            deltaPhaseColumn(row) = deltaPhase(row, 0); // 1st pol only
        }
        else if (row < deltaPhaseColumn.rows())
        {
            deltaPhaseColumn(row) = 0; // deltaPhaseColumn is of size deltaPhaseRows+1 where the last/extra row = 0
        }
    }
} // namespace cuda
} // namespace icrar
