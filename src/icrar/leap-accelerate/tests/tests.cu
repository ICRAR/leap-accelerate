
#include <hip/hip_runtime.h>
///**
// * ICRAR - International Centre for Radio Astronomy Research
// * (c) UWA - The University of Western Australia
// * Copyright by UWA(in the framework of the ICRAR)
// * All rights reserved
// *
// * This library is free software; you can redistribute it and/or
// * modify it under the terms of the GNU Lesser General Public
// * License as published by the Free Software Foundation; either
// * version 2.1 of the License, or (at your option) any later version.
// *
// * This library is distributed in the hope that it will be useful,
// * but WITHOUT ANY WARRANTY; without even the implied warranty of
// * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.See the GNU
// * Lesser General Public License for more details.
// *
// * You should have received a copy of the GNU Lesser General Public
// * License along with this library; if not, write to the Free Software
// * Foundation, Inc., 59 Temple Place, Suite 330, Boston,
// * MA 02111 - 1307  USA
// */
//
//#include <icrar/leap-accelerate/math/vector.cuh>
//
//#include <gtest/gtest.h>
//
//#include <stdio.h>
//
//#include <cuda_runtime.h>
//#include <device_launch_parameters.h>
//
//class compute_tests : public testing::Test
//{
//public:
//    int test_add() {
//        int* a;
//        int* b;
//        int* out;
//        add<<<1, 1>>>(a, b, out);
//        return 0;
//    }
//};
//
//TEST_F(compute_tests, test_add) { test_add(); }
