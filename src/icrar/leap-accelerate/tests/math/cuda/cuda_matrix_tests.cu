/**
 * ICRAR - International Centre for Radio Astronomy Research
 * (c) UWA - The University of Western Australia
 * Copyright by UWA(in the framework of the ICRAR)
 * All rights reserved
 *
 * This library is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation; either
 * version 2.1 of the License, or (at your option) any later version.
 *
 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with this library; if not, write to the Free Software
 * Foundation, Inc., 59 Temple Place, Suite 330, Boston,
 * MA 02111 - 1307  USA
 */

#include <hip/hip_runtime.h>

#include <icrar/leap-accelerate/cuda/helper_cuda.cuh>
#include <icrar/leap-accelerate/math/cuda/matrix.cuh>
#include <icrar/leap-accelerate/math/cuda/vector.cuh>

#include <gtest/gtest.h>

#include <stdio.h>
#include <array>

class cuda_matrix_tests : public testing::Test
{
public:
    cuda_matrix_tests()
    {

    }

    void SetUp() override
    {
        int deviceCount = 0;
        checkCudaErrors(hipGetDeviceCount(&deviceCount));
        ASSERT_EQ(1, deviceCount);
        // See this page: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__DEVICE.html
    }

    void TearDown() override
    {

    }

    template<typename T>
    void test_matrix_add()
    {
        using MatrixXT = Eigen::Matrix<T, Eigen::Dynamic, Eigen::Dynamic>;

        auto a = MatrixXT(3,3);
        a << 1, 2, 3,
             4, 5, 6,
             7, 8, 9;

        auto b = a;
        auto c = MatrixXT(3,3); 

        icrar::cuda::h_add<T, -1, -1>(a, b, c);

        MatrixXT expected = a + b;
        ASSERT_EQ(c, expected);
    }

    template<typename T>
    void test_matrix_matrix_multiply()
    {
        using MatrixXT = Eigen::Matrix<T, -1, -1>;

        auto a = MatrixXT(3,3);
        a << 1, 0, 0,
             0, 1, 0,
             0, 0, 1;

        auto b = MatrixXT(3,3);
        b << 1, 0, 0,
             0, 1, 0,
             0, 0, 1;

        auto c = MatrixXT(3,3); 

        icrar::cuda::h_multiply(a, b, c);

        MatrixXT expected = a * b;

        //ASSERT_EQ(c, expected);
        ASSERT_EQ(c(0,0), 1);
        ASSERT_EQ(c(0,1), 0);
        ASSERT_EQ(c(0,2), 0);
        ASSERT_EQ(c(1,0), 0);
        ASSERT_EQ(c(1,1), 1);
        ASSERT_EQ(c(1,2), 0);
        ASSERT_EQ(c(2,0), 0);
        ASSERT_EQ(c(2,1), 0);
        ASSERT_EQ(c(2,2), 1);
    }

    template<typename T>
    void test_matrix_vector_multiply()
    {
        using MatrixXT = Eigen::Matrix<T, Eigen::Dynamic, Eigen::Dynamic>;

        auto a = MatrixXT(3,3);
        a << 1, 2, 3,
             4, 5, 6,
             7, 8, 9;

        auto b = Eigen::Matrix<T, Eigen::Dynamic, 1>(3, 1);
        auto c = Eigen::Matrix<T, Eigen::Dynamic, 1>(3, 1); 

        icrar::cuda::h_multiply(a, b, c);

        MatrixXT expected = a * b;
        ASSERT_EQ(c, expected);
    }

    template<typename T>
    void test_scalear_matrix_multiply()
    {

    }
};

TEST_F(cuda_matrix_tests, test_matrix_add) { test_matrix_add<double>(); }
TEST_F(cuda_matrix_tests, test_matrix_matrix_multiply) { test_matrix_matrix_multiply<double>(); }
TEST_F(cuda_matrix_tests, test_matrix_vector_multiply) { test_matrix_vector_multiply<double>(); }
TEST_F(cuda_matrix_tests, test_scalear_matrix_multiply) { test_scalear_matrix_multiply<double>(); }
